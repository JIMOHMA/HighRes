/****************************************************************
 * To run assignment, run make first and then use "hw5 radius inputfile.ppm outfile.ppm" 
 *
 * Read and write PPM files.  Only works for "raw" format.
 * To get timing of program, run command "time Editor brushSize inputfile.ppm outputfile.ppm"
 * here brushSize is the level of blurness you want for the image being edited, 
 * inputfile.ppm is the image being edited and outfile.ppm is the edited image. 
 *
 ****************************************************************/
// Developer's Name :: Ayodele Jimoh 
// src: ImageEditor.cu
// Date :: 12/04/2019


#include <hip/hip_runtime.h>
#include <omp.h>
#include <stdlib.h>
#include <stdio.h>
#include <ctype.h>
#include <time.h>

 typedef struct Image
{
	  int width;
	  int height;
	  unsigned char *data;
} Image;

// Thread block size
#define BLOCK_SIZE 32

/************************ private functions ***********************/
struct Image *originalImage; // pointer object of initial image to be read from file
int someValue = 2;
int *brushRadius = &someValue; // brush radius.... VALUE IS CHANGED WHEN READ FROM IMPUT... This was done to avoid seg fault

/* die gracelessly */
static void
__host__ die(char const *message)
{
  fprintf(stderr, "ppm: %s\n", message);
  exit(1);
}


/* check a dimension (width or height) from the image file for reasonability */
__host__ static void
checkDimension(int dim)
{
  if (dim < 1 || dim > 6000) 
	die("file contained unreasonable width or height");
}


/* read a header: verify format and get width and height */
__host__ static void
readPPMHeader(FILE *fp, int *width, int *height)
{
  char ch;
  int  maxval;

  if (fscanf(fp, "P%c\n", &ch) != 1 || ch != '6') 
	die("file is not in ppm raw format; cannot read");

  /* skip comments */
  ch = getc(fp);
  while (ch == '#')
	{
	  do {
	ch = getc(fp);
	  } while (ch != '\n');	/* read to the end of the line */
	  ch = getc(fp);            
	}

  if (!isdigit(ch)) die("cannot read header information from ppm file");

  ungetc(ch, fp);		/* put that digit back */

  /* read the width, height, and maximum value for a pixel */
  fscanf(fp, "%d%d%d\n", width, height, &maxval);

  if (maxval != 255) die("image is not true-color (24 bit); read failed");
  
  checkDimension(*width);
  checkDimension(*height);
}

/************************ exported functions ****************************/

__host__ Image *ImageCreate(int width, int height)
{
  Image *image = (Image *) malloc(sizeof(Image));

  if (!image) die("cannot allocate memory for new image");

  image->width  = width;
  image->height = height;
  image->data   = (unsigned char *) malloc(width * height * 3);

  if (!image->data) die("cannot allocate memory for new image");

  return image;
}
  

__host__ Image *ImageRead(char const *filename)
{
  int width, height, num, size;

  Image *image = (Image *) malloc(sizeof(Image));
  FILE  *fp    = fopen(filename, "rb");

  if (!image) die("cannot allocate memory for new image");
  if (!fp)    die("cannot open file for reading");

  readPPMHeader(fp, &width, &height);

  size          = width * height * 3;
  image->data   = (unsigned  char*) malloc(size);
  image->width  = width;
  image->height = height;

  if (!image->data) die("cannot allocate memory for new image");

  num = fread((void *) image->data, 1, (size_t) size, fp);

  if (num != size) die("cannot read image data from file");

  fclose(fp);

  return image;
}


__host__ void ImageWrite(Image *image, char const *filename)
{
  int num;
  int size = image->width * image->height * 3;

  FILE *fp = fopen(filename, "wb");

  if (!fp) die("cannot open file for writing");

  fprintf(fp, "P6\n%d %d\n%d\n", image->width, image->height, 255);

  num = fwrite((void *) image->data, 1, (size_t) size, fp);

  if (num != size) die("cannot write image data to file");

  fclose(fp);
}  


__host__ int ImageWidth(Image *image)
{
  return image->width;
}


__host__ int ImageHeight(Image *image)
{
  return image->height;
}

// This function is not utilized anywhere in this code, so we put it on host
// since it has no application in any parallelization eoperation. Just for convenience. 
__host__ void   
ImageClear(Image *image, unsigned char red, unsigned char green, unsigned char blue)
{
  int i;
  int pix = image->width * image->height;

  unsigned char *data = image->data;

  for (i = 0; i < pix; i++)
	{
	  *data++ = red;
	  *data++ = green;
	  *data++ = blue;
	}
}

__device__ void ImageSetPixel(unsigned char *d_dataOut, int x, int y, int chan, unsigned char val, int width)
{
  int offset = (y * width + x) * 3 + chan;

  d_dataOut[offset] = val;
}

__device__ unsigned  char ImageGetPixel(unsigned char *d_dataIn, int x, int y, int chan, int width)
{
  int offset = (y * width + x) * 3 + chan;

  return d_dataIn[offset];
}

// sets the minX value for a pixel
__device__ int setMinX(int x, int *d_brushRadius){
	int minX;
	if(x - *d_brushRadius < 0){ // this is the case whereby the minX value exceeds the boundary
		minX = 0;
	}
	else{
		minX = x - *d_brushRadius;
	}
	return minX;
}
// sets the minY value for a pixel 
__device__ int setMinY(int y, int *d_brushRadius){

	int minY;
	if(y - *d_brushRadius < 0){ // this is the case whereby the minY value exceeds the boundary
		minY = 0;
	}
	else{
		minY = y - *d_brushRadius;
	}
	return minY;
}
// sets the maxX value for a pixel 
__device__ int setMaxX(int x, int *d_brushRadius, int width){
	int maxX;
	if(x + *d_brushRadius > width){ // this is the case whereby the maxX value exceeds the boundary
		maxX = width;
	}
	else{
		maxX = x + *d_brushRadius;
	}
	return maxX;
}
// sets the maxY value for a pixel 
__device__ int setMaxY(int y, int *d_brushRadius, int height){
	int maxY;
	if(y + *d_brushRadius > height){ // this is the case whereby the maxY value exceeds the boundary
		maxY = height;
	}
	else{
		maxY = y + *d_brushRadius;
	}
	return maxY;
}


// gets the red channel for a pixel in an image 
// this is a gpu fucntion because the function is 
// being usig by a kernel on the GPU side
__device__ int getRedPixel(int minX, int minY, int maxX, int maxY, unsigned char *d_dataIn, int width){
	int temp = 0;
	int i;
	int j;
	for (i = minX; i<maxX; i++){
		for (j = minY; j<maxY; j++){
			temp += ImageGetPixel(d_dataIn, i, j, 0, width);
		}
	}
	return temp; // temp is the calculated channel value being returned
}

// gets the green channel for a pixel in an image 
// this is a gpu fucntion because the function is 
// being usig by a kernel on the GPU side
__device__ int getGreenPixel(int minX, int minY, int maxX, int maxY, unsigned char *d_dataIn, int width){
	int temp = 0;
	int i;
	int j;
	for (i = minX; i<maxX; i++){
		for (j = minY; j<maxY; j++){
			temp += ImageGetPixel(d_dataIn, i, j, 1, width);
		}
	}
	return temp;
}

// gets the blue channel for a pixel in an image 
// this is a gpu fucntion because the function is 
// being usig by a kernel on the GPU side
__device__ int getBluePixel(int minX, int minY, int maxX, int maxY, unsigned char *d_dataIn, int width){

	int temp = 0;
	int i;
	int j;
	for (i = minX; i<maxX; i++){
		for (j = minY; j<maxY; j++){
			temp += ImageGetPixel(d_dataIn, i, j, 2, width);
		}
	}
	return temp;
}

// function for applying the filter function to the image
// Function to be parallelized which is callable from the host CPU. 
// remember to change d_inputImage d_dataIn, and d_outputImage to d_dataOut within this function
// Remember d_dataOut holds the data for the filtered image
__global__ void processImage(unsigned char *d_dataIn, unsigned char *d_dataOut, int width, int height, int *d_brushRadius){ 
	
	// Need to define the blockIdx and threadIdx
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	//printf("x and y are %d, %d\n", x, y);
	// Variable holder for the colour channels
	int pr = 0;
	int pg = 0;
	int pb = 0;
	int minX, minY, maxX, maxY;
	int numPixels;

	// printf("1) Value of brush is %d\n", *d_brushRadius);

	// we need to set the max and min values for every pixel we try to process
	// device function to set the min and max values of the boundary
	minX = setMinX(x, d_brushRadius);
	minY = setMinY(y, d_brushRadius);
	maxX = setMaxX(x, d_brushRadius, width);
	maxY = setMaxY(y, d_brushRadius, height);

	numPixels = (maxY - minY) * (maxX - minX);

	// these three functions compute the the unsigned car value for 
	// a channel for a given radius brush buy considering all the pixels 
	// withing the radius from the point (x,y)
	pr = getRedPixel(minX, minY, maxX, maxY, d_dataIn, width); // d_inputImage should be replaced with d_dataIn
	pg = getGreenPixel(minX, minY, maxX, maxY, d_dataIn, width);
	pb = getBluePixel(minX, minY, maxX, maxY, d_dataIn, width);

	pr = pr/numPixels;
	pg = pg/numPixels;
	pb = pb/numPixels;
	// sets the values of the channels in the newImage being processed for 
	// filtering. 
	ImageSetPixel(d_dataOut, x, y, 0, pr, width); // d_outputImage should be replaced with d_dataOut
	ImageSetPixel(d_dataOut, x, y, 1, pg, width);
	ImageSetPixel(d_dataOut, x, y, 2, pb, width);

}

int main(int argc, char* argv[]){
	if (argc != 4){
		printf("Incorrect number of command line argument ######!\n");
		exit(0);
	}

	// printf("The value of brushSize is: %d\n", brushRadius);
	// printf("The value of brushSize after assignment is: %d\n", radius);
	// printf("The value of brushSize after assignment is: %d\n", *brushRadius);
	
	int radius = atoi(argv[1]); // our brush radius
	*brushRadius = radius;
	char *inputFile = argv[2];  // input fileName containing our image 
	char *outputFile = argv[3]; // output fileName for which we'll be writing out processed image into 
	originalImage = ImageRead(inputFile);
	int width = ImageWidth(originalImage);
	int height = ImageHeight(originalImage);

	struct Image *h_inputImage = ImageRead(inputFile); // pointer object for the new image to be outputted on host machine

	// structure that will eventually hold the output/edited 
	// image on the host machine. 
	// Create am empty image inside h_outputImage
	struct Image *h_outputImage = ImageCreate(width, height);
	// unsigned char *h_dataOut = NULL; //h_outputImage->data; // we've copied the empty data in h_outputImage to h_dataOut

	printf("Image width is %d, and height is %d\n", width, height);
	printf("NewImage width is %d, and height is %d\n", ImageWidth(h_inputImage), h_inputImage->height);

	// pointers for holding out input image data and output image data
	unsigned char *d_dataIn = NULL;
	unsigned char *d_dataOut = NULL;
	int *d_brushRadius = NULL; // pointer for radius brush

	// size of the memory we about to allocate for image on device
	size_t sizeData = h_inputImage->width * h_inputImage->height * 3;
	size_t brushSize = sizeof(int);

	// Allocate memory on the device for the d_inputImage and d_outputImage
	hipMalloc(&d_dataIn, sizeData);
	hipMalloc(&d_dataOut, sizeData);
	hipMalloc(&d_brushRadius, brushSize);

	// Copy data host image memory into device GPU image memory
	hipMemcpy(d_dataIn, h_inputImage->data, sizeData, hipMemcpyHostToDevice);
	hipMemcpy(d_brushRadius, brushRadius, brushSize, hipMemcpyHostToDevice);
	// printf("Begin time ...\n");
	// time_t begin, end;
	// time(&begin);

	// Launch Kernel for parallelization
	// This is the kernel function that we want to parallelize for 
	// faster rendering of filter to image. 
	// Note: Add image as an argument which is a pointer so any update 
	// we do is gonna be done to this outputImage for the device machine i.e d_outputImage
	// TO DO: find out how to allocate the blocks and threads for the kernel
	dim3 dimBlock(32,32); // 32 threads in each block of threads
	dim3 dimGrid(width/dimBlock.x, height/dimBlock.y); 
	processImage<<<dimGrid, dimBlock>>>(d_dataIn, d_dataOut, width, height, d_brushRadius);
	// End of launching kernel
	hipDeviceSynchronize();

	// Let's copy out final image data back onto the CPU
	hipMemcpy(h_outputImage->data, d_dataOut, sizeData, hipMemcpyDeviceToHost);

	// write final/edited image data into output file
	ImageWrite(h_outputImage, outputFile); // writes the final edited image into the outputFile. 
	// time(&end); // end timer for Image Processing. 
	// double time_spent = difftime(end, begin); // time taken to process image.
	// printf("Time taken is %f seconds.\n", time_spent);

	// Free the allocated memory on the GPU 
	hipFree(d_dataIn);
	hipFree(d_dataOut);
	hipFree(d_brushRadius);

	return 0;
}